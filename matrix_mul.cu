#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include<random>
#include<cassert>
#include<iostream>
#include"time.h"

using namespace std;


__global__ void matrixMul(int *a, int *b, int *c, int N) {
	
	//calcular o tamanho de cada thread
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N && col < N)
	{
		int tmp = 0;

		for (int i = 0; i < N; i++)
		{
			tmp += a[row * N + i] * b[i * N + col];
		}

		c[row * N + col] = tmp;
	}
	
}


//inicializar as matrizes com numeros aletorios
void init_matrix(int* m, int N)
{
	for (int i = 0; i < N * N; i++)
	{
		m[i] = rand() % 100;
	}
}


//CPU
void cpu_result(int* a, int* b, int* c, int N)
{
	
	int tmp;
	for (int i = 0; i < N; i++)//row
	{
		for (int j = 0; j < N; j++)//cpl
		{
			tmp = 0;
			for (int k = 0; k < N; k++)//elemento no row-col
			{
				tmp += a[i * N + k] * b[k * N + j];
			}

			//check results
			//assert(tmp == c[i * N + j]);
			c[i * N + j] = tmp;
		}
	}
	
}
int main() {
	//tamanho da matriz
	int N = 1 << 10;
	size_t bytes = N * N * sizeof(int);

	Timer t1, t2;
	double timer1, timer2;
	//alocar memoria pras matrizes
	int* a, *b,  *c;
	hipMallocManaged(&a, bytes);
	hipMallocManaged(&b, bytes);
	hipMallocManaged(&c, bytes);

	//inicializar as matrizes
	init_matrix(a, N);
	init_matrix(b, N);

	int threads = 16;
	int blocks = (N + threads - 1) / threads;

	//setup kernel parameters
	dim3 THREADS(threads, threads);
	dim3 BLOCKS(blocks, blocks);
	t1.reset();
	t1.start();
	matrixMul << <BLOCKS, THREADS >> > (a, b, c, N);
	hipDeviceSynchronize();
	t1.finish();

	t2.reset();
	t2.start();
	cpu_result(a, b, c, N);
	t2.finish();

	cout << "PROGRAM COMPLETED" << endl;
	timer1 = t1.getElapsedTimeMs();
	timer2 = t2.getElapsedTimeMs();
	cout << "Timer GPU: " << timer1 << endl << "Timer CPU: " << timer2 << endl;
	return 0;

}